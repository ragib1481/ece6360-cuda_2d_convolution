#include "hip/hip_runtime.h"
//
// Created by ragib1481 on 10/22/22.
//

#include "Image.cuh"

Image::Image(thrust::host_vector<char> &img, short width, short height) {

    this->width = width;
    this->height = height;
    image.resize(width * height);
    for (size_t i=0; i < width * height; i++) {
        image[i].b = ((float)(unsigned char)img[3 * i]) / 255.0;
        image[i].g = ((float)(unsigned char)img[3 * i + 1]) / 255.0;
        image[i].r = ((float)(unsigned char)img[3 * i + 2]) / 255.0;
    }
}

Image::Image(thrust::host_vector<pixel> &img, short width, short height) {
    this->image = img;
}

pixel* Image::getPointer() {
    return &image[0];
}

void Image::toBytes(thrust::host_vector<char>& bytes, bool scale) {
    bytes.resize(image.size() * 3);

    float max = 0.0;
    if (scale) {
        for (size_t i = 0; i < image.size(); i++) {
            if (image[i].b > max)
                max = image[i].b;
            if (image[i].g > max)
                max = image[i].g;
            if (image[i].r > max)
                max = image[i].r;
        }
    }

    for (size_t i = 0; i < image.size(); i++){
        if (scale) {
            bytes[3 * i] = (char) (255 * image[i].b / max);
            bytes[3 * i + 1] = (char) (255 * image[i].g / max);
            bytes[3 * i + 2] = (char) (255 * image[i].r / max);
        }
        else{
            bytes[3 * i] = (char) (image[i].b * 255.0);
            bytes[3 * i + 1] = (char) (image[i].g * 255.0);
            bytes[3 * i + 2] = (char) (image[i].r * 255.0);
        }
    }
}

thrust::host_vector<pixel> Image::getImage() {
    return image;
}
