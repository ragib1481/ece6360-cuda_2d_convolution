#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <cmath>
#include <chrono>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "Image.cuh"
#include "FileHandler.cuh"


using namespace std;
typedef std::chrono::high_resolution_clock timer;

thrust::host_vector<pixel> checkCalcError(const thrust::host_vector<pixel> cpuOut, const thrust::host_vector<pixel> gpuOut) {
    if (cpuOut.size() != gpuOut.size()) {
        cout << "Size mismatch" << endl;
        return thrust::host_vector<pixel>();
    }
    thrust::host_vector<pixel> errors(cpuOut.size());
    float error = 0.0;
    for (size_t i = 0; i < cpuOut.size(); i++) {
        error += abs(cpuOut[i].r - gpuOut[i].r);
        error += abs(cpuOut[i].g - gpuOut[i].g);
        error += abs(cpuOut[i].b - gpuOut[i].b);
    }
    cout << "Total absolute error: " << error << endl;
    return errors;
}


void getGaussianFilter(thrust::host_vector<float>& filter, int sigma) {
    /* This function returns a vector containing a 1d gaussian filter assuming 0 mean.
     * Since we are assuming the filter is symmetric, this is a more
     * efficient approach instead of using a 2d gaussian filter.
     */
    int k = 6 * sigma + 1;
    float sig = (float) sigma;
    float x;

    filter.resize(k);
    for (int i = 0; i < filter.size(); i++) {
        x = (float) (i - k / 2);
        filter[i] = (float)(exp(-1.0 * x * x / (2.0 * sig * sig)) / sqrt(2.0 * sig * sig * M_PI));
        //filter[i] = 1.0 / (float)(256);
    }
}


void convolveHeight(thrust::host_vector<pixel>& out, const pixel* sig, const float* filter, short& width, short& height, int filterSize) {
    short newHeight = height - filterSize + 1;
    float resultR;
    float resultG;
    float resultB;

    for (int i = 0; i < width; i++) {
        for (int j = 0; j < newHeight; j++) {
            resultR = 0.0;
            resultG = 0.0;
            resultB = 0.0;
            for (int k = 0; k < filterSize; k++) {
                resultR += sig[i + (j + k) * width].r * filter[k];
                resultG += sig[i + (j + k) * width].g * filter[k];
                resultB += sig[i + (j + k) * width].b * filter[k];
            }
            out[i + j * width].r = resultR;
            out[i + j * width].g = resultG;
            out[i + j * width].b = resultB;
        }
    }
    height = newHeight;
}


void convolveWidth(thrust::host_vector<pixel>& out, const pixel* sig, const float* filter, short& width, short& height, int filterSize) {
    short newWidth = width - filterSize + 1;
    float resultR;
    float resultG;
    float resultB;

    for (int i = 0; i < height; i++) {
        for (int j = 0; j < newWidth; j++) {
            resultR = 0.0;
            resultG = 0.0;
            resultB = 0.0;
            for (int k = 0; k < filterSize; k++) {
                resultR += sig[i * width + j + k].r * filter[k];
                resultG += sig[i * width + j + k].g * filter[k];
                resultB += sig[i * width + j + k].b * filter[k];
            }
            out[i * newWidth + j].r = resultR;
            out[i * newWidth + j].g = resultG;
            out[i * newWidth + j].b = resultB;
        }
    }
    width = newWidth;
}


////************************************** GPU implementations ********************************************
__constant__ float filterConstMem[65536 / sizeof(float)];
__global__
void convolveHeightKernel(pixel* out, const pixel* sig, const short width,
                          const short height, const short newHeight, const int filterSize) {
    extern __shared__ pixel signalSharedMem[];                                // pointer to the shared memory
    size_t ix = blockDim.x * blockIdx.x + threadIdx.x;
    size_t iy = blockDim.y * blockIdx.y + threadIdx.y;
    size_t idx;
    int numElementToCopy = blockDim.y + filterSize - 1;

    float r = 0.0;
    float g = 0.0;
    float b = 0.0;

    // copy signal to the shared memory
    for (int i = threadIdx.y; (i < numElementToCopy) && ((blockDim.y * blockIdx.y + i) < height); i += blockDim.y) {
        signalSharedMem[i * blockDim.x + threadIdx.x] = sig[ix + (blockDim.y * blockIdx.y + i) * width];
    }
    __syncthreads();

    if ((ix < width) && (iy < newHeight)) {

        for (int k = 0; k < filterSize; k++) {
            //idx = ix + (iy + k) * width;
            //r += sig[idx].r * filterConstMem[k];
            //g += sig[idx].g * filterConstMem[k];
            //b += sig[idx].b * filterConstMem[k];
            idx = (threadIdx.y + k) * blockDim.x + threadIdx.x;
            r += signalSharedMem[idx].r * filterConstMem[k];
            g += signalSharedMem[idx].g * filterConstMem[k];
            b += signalSharedMem[idx].b * filterConstMem[k];
        }
        idx = ix + iy * width;
        out[idx].r = r;
        out[idx].g = g;
        out[idx].b = b;
    }
}

__global__
void convolveWidthKernel(pixel* out, const pixel* sig, const short width,
                         const short newWidth, const short height, const int filterSize) {
    extern __shared__ pixel signalSharedMem[];                                  // pointer to the shared memory
    size_t ix = blockDim.x * blockIdx.x + threadIdx.x;                          // calculate index along x dimension
    size_t iy = blockDim.y * blockIdx.y + threadIdx.y;                          // calculate index along y dimension
    size_t idx;                                                                 // temporary variable to hold index
    int numElementToCopy = blockDim.x + filterSize - 1;

    float r = 0.0;
    float g = 0.0;
    float b = 0.0;

    // copy signal to the shared memory
    for (int i = threadIdx.x; (i < numElementToCopy) && ((blockDim.x * blockIdx.x + i) < width); i += blockDim.x) {
        signalSharedMem[threadIdx.y * numElementToCopy + i] = sig[iy * width + blockIdx.x * blockDim.x + i];
    }
    __syncthreads();

    // check if the indices are out scope
    if ((ix < newWidth) && (iy < height)) {

        // perform convolution
        for (int k = 0; k < filterSize; k++) {
            //idx = iy * width + ix + k;
            //r += sig[idx].r * filterConstMem[k];
            //g += sig[idx].g * filterConstMem[k];
            //b += sig[idx].b * filterConstMem[k];
            idx = threadIdx.y * numElementToCopy + threadIdx.x + k;
            r += signalSharedMem[idx].r * filterConstMem[k];
            g += signalSharedMem[idx].g * filterConstMem[k];
            b += signalSharedMem[idx].b * filterConstMem[k];
        }

        // put data into the output
        idx = iy * newWidth + ix;
        out[idx].r = r;
        out[idx].g = g;
        out[idx].b = b;
    }
}

thrust::host_vector<pixel> convolve2dGpu(const thrust::host_vector<pixel> sig, const thrust::host_vector<float>filter,
                                         short& width, short& height, const int filterSize, FileHandler& handler){
    short newWidth, newHeight;

    ////********************************** perform convolution along the width ****************************
    //// define device variables for convolution along the width
    newWidth = width - filterSize + 1;
    thrust::device_vector<pixel> out1Gpu(newWidth * height);
    pixel* out1Ptr = thrust::raw_pointer_cast(out1Gpu.data());

    thrust::device_vector<pixel> in1 = sig;
    pixel* in1Ptr = thrust::raw_pointer_cast(in1.data());

    thrust::device_vector<float> convFilterGpu = filter;
    float* filterPtr = thrust::raw_pointer_cast(convFilterGpu.data());

    //// define filter launch parameters for convolution along the width
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, 0);
    dim3 blockDimension(32, 16, 1);
    dim3 gridDimension((newWidth + blockDimension.x - 1) / blockDimension.x,
                       (height + blockDimension.y - 1) / blockDimension.y, 1);

    size_t sharedBytes = blockDimension.y * (blockDimension.x + filterSize - 1) * sizeof(pixel);

    // shared memory size
    if (sharedBytes > devProp.sharedMemPerBlock) {
        cout << "Invalid filter size." << endl;
        thrust::host_vector<pixel> out((width - filterSize + 1) * (height - filterSize + 1));
        for (size_t i = 0; i < out.size(); i++){
            out[i].r = 0.0; out[i].g = 0.0; out[i].b = 0.0;
        }
        return out;
    }

    // copy filter to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(filterConstMem), filterPtr, filterSize * sizeof(float), 0, hipMemcpyHostToDevice);

    convolveWidthKernel<<<gridDimension, blockDimension, sharedBytes>>>
                            (out1Ptr, in1Ptr, width, newWidth, height, filterSize);
    hipDeviceSynchronize();
    width = newWidth;

    ////********************************** perform convolution along the height ***************************
    //sharedBytes = filterSize * sizeof(float);
    blockDimension.y = 8;
    sharedBytes = blockDimension.x * (blockDimension.y + filterSize - 1) * sizeof(pixel);
    if (sharedBytes > devProp.sharedMemPerBlock) {
        cout << "Invalid filter size." << endl;
        thrust::host_vector<pixel> out((width - filterSize + 1) * (height - filterSize + 1));
        for (size_t i = 0; i < out.size(); i++){
            out[i].r = 0.0; out[i].g = 0.0; out[i].b = 0.0;
        }
        return out;
    }
    newHeight = height - filterSize + 1;
    gridDimension.y = (newHeight + blockDimension.y - 1) / blockDimension.y;

    thrust::device_vector<pixel> out2Gpu(newWidth * newHeight);
    pixel* out2Ptr = thrust::raw_pointer_cast(out2Gpu.data());
    convolveHeightKernel<<<gridDimension, blockDimension, sharedBytes>>>
                            (out2Ptr, out1Ptr, width, height, newHeight, filterSize);
    hipDeviceSynchronize();
    height = newHeight;

    thrust::host_vector<pixel> outHost = out2Gpu;

    return outHost;
}


int main(int argc, char* argv[]) {
    ////********************************** parse command line arguments **********************************
    if (argc != 3)
        return 1;
    string filename(argv[1]);
    int sigma = atoi(argv[2]);

    ////********************************** declare variables *********************************************
    FileHandler handler;
    short width;
    short height;

    ////********************************** generate gaussian filter **************************************
    thrust::host_vector<float> filter;
    getGaussianFilter(filter, sigma);
    float* kernelPtr = &filter[0];

    ////********************************** load image ****************************************************
    thrust::host_vector<char> imageRaw = handler.loadImage(filename, width, height);
    const short widthInit = width;
    const short heightInit = height;
    Image image(imageRaw, width, height);
    pixel* imagePtr = image.getPointer();

    //////********************************** perform convolution(CPU) **************************************
    auto start = timer::now();
    thrust::host_vector<pixel> out1Cpu;
    out1Cpu.resize((width - filter.size() + 1) * height);
    convolveWidth(out1Cpu, imagePtr, kernelPtr, width, height, filter.size());

    thrust::host_vector<pixel> out2Cpu;
    out2Cpu.resize(width * (height - filter.size() + 1));
    convolveHeight(out2Cpu, &out1Cpu[0], kernelPtr, width, height, filter.size());
    auto end = timer::now();

    // report computation time
    std::chrono::milliseconds t = std::chrono::duration_cast<std::chrono::milliseconds> (end - start);
    cout << "CPU Elapsed time: " << t.count() << "ms" << endl;

    //////********************************** save data (CPU) *********************************************
    Image outputCpu(out2Cpu, width, height);
    thrust::host_vector<char> bytesCpu;
    outputCpu.toBytes(bytesCpu, true);
    handler.saveImage(bytesCpu, "./resultCpu.tga", width, height);

    ////********************************** perform convolution(GPU) **************************************
    width = widthInit;
    height = heightInit;
    start = timer::now();
    thrust::host_vector<pixel> outGpu = convolve2dGpu(image.getImage(), filter, width, height, filter.size(), handler) ;
    end = timer::now();

    t = std::chrono::duration_cast<std::chrono::milliseconds> (end - start);
    cout << "GPU Elapsed time: " << t.count() << "ms" << endl;

    //////********************************** save data (GPU) *********************************************
    Image outputGpu(outGpu, width, height);
    thrust::host_vector<char> bytesGpu;
    outputGpu.toBytes(bytesGpu, true);
    handler.saveImage(bytesGpu, "./resultGpu.tga", width, height);

    //////********************************** save data (err) *********************************************
    checkCalcError(out2Cpu, outGpu);
    return 0;
}
