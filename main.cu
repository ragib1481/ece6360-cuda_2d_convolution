#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <cmath>
#include <chrono>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "Image.cuh"
#include "FileHandler.cuh"


using namespace std;
typedef std::chrono::high_resolution_clock timer;

void checkCalcError(const thrust::host_vector<pixel> cpuOut, const thrust::host_vector<pixel> gpuOut) {
    if (cpuOut.size() != gpuOut.size()) {
        cout << "Size mismatch" << endl;
        return;
    }
    double error = 0.0;
    for (size_t i = 0; i < cpuOut.size(); i++) {
        error += abs(cpuOut[i].r - gpuOut[i].r);
        error += abs(cpuOut[i].g - gpuOut[i].g);
        error += abs(cpuOut[i].b - gpuOut[i].b);
    }
    cout << "Total absolute error: " << error << endl;
}


void getGaussianFilter(thrust::host_vector<double>& kernel, int sigma) {
    /* This function returns a vector containing a 1d gaussian kernel assuming 0 mean.
     * Since we are assuming the kernel is symmetric, this is a more
     * efficient approach instead of using a 2d gaussian kernel.
     */
    int k = 6 * sigma + 1;
    double sig = (double) sigma;
    double x;

    kernel.resize(k);
    for (int i = 0; i < kernel.size(); i++) {
        x = (double) (i - k / 2);
        kernel[i] = (double)(exp(-1.0 * x * x / (2.0 * sig * sig)) / sqrt(2.0 * sig * sig * M_PI));
        //kernel[i] = 1.0/(double)k;
    }
}


void convolveHeight(thrust::host_vector<pixel>& out, const pixel* sig, const double* filter, short& width, short& height, int kernelSize) {
    short newHeight = height - kernelSize + 1;
    double resultR;
    double resultG;
    double resultB;

    for (int i = 0; i < width; i++) {
        for (int j = 0; j < newHeight; j++) {
            resultR = 0.0;
            resultG = 0.0;
            resultB = 0.0;
            for (int k = 0; k < kernelSize; k++) {
                resultR += sig[i + (j + k) * width].r * filter[k];
                resultG += sig[i + (j + k) * width].g * filter[k];
                resultB += sig[i + (j + k) * width].b * filter[k];
            }
            out[i + j * width].r = resultR;
            out[i + j * width].g = resultG;
            out[i + j * width].b = resultB;
        }
    }
    height = newHeight;
}


void convolveWidth(thrust::host_vector<pixel>& out, const pixel* sig, const double* filter, short& width, short& height, int kernelSize) {
    short newWidth = width - kernelSize + 1;
    double resultR;
    double resultG;
    double resultB;

    for (int i = 0; i < height; i++) {
        for (int j = 0; j < newWidth; j++) {
            resultR = 0.0;
            resultG = 0.0;
            resultB = 0.0;
            for (int k = 0; k < kernelSize; k++) {
                resultR += sig[i * width + j + k].r * filter[k];
                resultG += sig[i * width + j + k].g * filter[k];
                resultB += sig[i * width + j + k].b * filter[k];
            }
            out[i * newWidth + j].r = resultR;
            out[i * newWidth + j].g = resultG;
            out[i * newWidth + j].b = resultB;
        }
    }
    width = newWidth;
}


////************************************** GPU implementations ********************************************
__global__
void convolveHeightKernel(pixel* out, const pixel* sig, const double* filter, const short width,
                          const short height, const short newHeight, const int kernelSize) {
    size_t ix = blockDim.x * blockIdx.x + threadIdx.x;
    size_t iy = blockDim.y * blockIdx.y + threadIdx.y;
    double r = 0.0;
    double g = 0.0;
    double b = 0.0;
    size_t idx;

    if ((ix < width) && (iy < newHeight)) {
        for (int k = 0; k < kernelSize; k++) {
            idx = ix + (iy + k) * width;
            r += sig[idx].r * filter[k];
            g += sig[idx].g * filter[k];
            b += sig[idx].b * filter[k];
        }
        idx = ix + iy * width;
        out[idx].r = r;
        out[idx].g = g;
        out[idx].b = b;
    }
}

__global__
void convolveWidthKernel(pixel* out, const pixel* sig, const double* filter, const short width,
                         const short newWidth, const short height, const int kernelSize) {
    size_t ix = blockDim.x * blockIdx.x + threadIdx.x;
    size_t iy = blockDim.y * blockIdx.y + threadIdx.y;
    double r = 0.0;
    double g = 0.0;
    double b = 0.0;
    size_t idx;

    if ((ix < newWidth) && (iy < height)) {

        for (int k = 0; k < kernelSize; k++) {
            idx = iy * width + ix + k;
            r += sig[idx].r * filter[k];
            g += sig[idx].g * filter[k];
            b += sig[idx].b * filter[k];
        }
        idx = iy * newWidth + ix;
        out[idx].r = r;
        out[idx].g = g;
        out[idx].b = b;
    }
}

thrust::host_vector<pixel> convolve2dGpu(const thrust::host_vector<pixel> sig, const thrust::host_vector<double>filter,
                                         short& width, short& height, int kernelSize){
    short newWidth, newHeight;

    ////********************************** perform convolution along the width ****************************
    //// define device variables for convolution along the width
    newWidth = width - kernelSize + 1;
    thrust::device_vector<pixel> out1Gpu(newWidth * height);
    pixel* out1Ptr = thrust::raw_pointer_cast(out1Gpu.data());

    thrust::device_vector<pixel> in1 = sig;
    pixel* in1Ptr = thrust::raw_pointer_cast(in1.data());

    thrust::device_vector<double> convKernelGpu = filter;
    double* filterPtr = thrust::raw_pointer_cast(convKernelGpu.data());

    //// define filter launch parameters for convolution along the width
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, 0);
    dim3 threads(devProp.maxThreadsPerBlock/32, 32);
    dim3 blocks(newWidth/threads.x + 1, height/threads.y + 1);

    convolveWidthKernel<<<blocks, threads>>>(out1Ptr, in1Ptr, filterPtr, width, newWidth, height, kernelSize);
    hipDeviceSynchronize();
    width = newWidth;

    ////********************************** perform convolution along the height ***************************
    newHeight = height - kernelSize + 1;
    blocks.y = newHeight/threads.y + 1;

    thrust::device_vector<pixel> out2Gpu(newWidth * newHeight);
    pixel* out2Ptr = thrust::raw_pointer_cast(out2Gpu.data());
    convolveHeightKernel<<<blocks, threads>>>(out2Ptr, out1Ptr, filterPtr, width, height, newHeight, kernelSize);
    hipDeviceSynchronize();
    height = newHeight;

    thrust::host_vector<pixel> outHost = out2Gpu;

    return outHost;
}


int main(int argc, char* argv[]) {
    ////********************************** parse command line arguments **********************************
    if (argc != 3)
        return 1;
    string filename(argv[1]);
    int sigma = atoi(argv[2]);

    ////********************************** declare variables *********************************************
    FileHandler handler;
    short width;
    short height;

    ////********************************** generate gaussian filter **************************************
    thrust::host_vector<double> filter;
    getGaussianFilter(filter, sigma);
    double* kernelPtr = &filter[0];

    ////********************************** load image ****************************************************
    thrust::host_vector<char> imageRaw = handler.loadImage(filename, width, height);
    const short widthInit = width;
    const short heightInit = height;
    Image image(imageRaw, width, height);
    pixel* imagePtr = image.getPointer();

    //////********************************** perform convolution(CPU) **************************************
    auto start = timer::now();
    thrust::host_vector<pixel> out1Cpu;
    out1Cpu.resize((width - filter.size() + 1) * height);
    convolveWidth(out1Cpu, imagePtr, kernelPtr, width, height, filter.size());

    thrust::host_vector<pixel> out2Cpu;
    out2Cpu.resize(width * (height - filter.size() + 1));
    convolveHeight(out2Cpu, &out1Cpu[0], kernelPtr, width, height, filter.size());
    auto end = timer::now();

    // report computation time
    std::chrono::milliseconds t = std::chrono::duration_cast<std::chrono::milliseconds> (end - start);
    cout << "CPU Elapsed time: " << t.count() << "ms" << endl;

    //////********************************** save data (CPU) *********************************************
    Image output(out2Cpu, width, height);
    thrust::host_vector<char> bytes;
    output.toBytes(bytes);
    handler.saveImage(bytes, "./resultCpu.tga", width, height);

    ////********************************** perform convolution(GPU) **************************************
    width = widthInit;
    height = heightInit;
    start = timer::now();
    thrust::host_vector<pixel> outGpu = convolve2dGpu(image.getImage(), filter, width, height, filter.size()) ;
    end = timer::now();

    t = std::chrono::duration_cast<std::chrono::milliseconds> (end - start);
    cout << "GPU Elapsed time: " << t.count() << "ms" << endl;

    //////********************************** save data (GPU) *********************************************
    Image outputGpu(outGpu, width, height);
    thrust::host_vector<char> bytesGpu;
    output.toBytes(bytesGpu);
    handler.saveImage(bytesGpu, "./resultGpu.tga", width, height);

    checkCalcError(out2Cpu, outGpu);

    return 0;
}
