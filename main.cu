#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <cmath>
#include <chrono>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "Image.cuh"
#include "FileHandler.cuh"


using namespace std;
typedef std::chrono::high_resolution_clock timer;

thrust::host_vector<pixel> checkCalcError(const thrust::host_vector<pixel> cpuOut, const thrust::host_vector<pixel> gpuOut) {
    if (cpuOut.size() != gpuOut.size()) {
        cout << "Size mismatch" << endl;
        return thrust::host_vector<pixel>();
    }
    thrust::host_vector<pixel> errors(cpuOut.size());
    float error = 0.0;
    for (size_t i = 0; i < cpuOut.size(); i++) {
        errors[i].r = abs(cpuOut[i].r - gpuOut[i].r);
        errors[i].g = abs(cpuOut[i].g - gpuOut[i].g);
        errors[i].b = abs(cpuOut[i].b - gpuOut[i].b);
        //error += (errors[i].r + errors[i].g + errors[i].b);
        error += abs(cpuOut[i].r - gpuOut[i].r);
        error += abs(cpuOut[i].g - gpuOut[i].g);
        error += abs(cpuOut[i].b - gpuOut[i].b);
    }
    cout << "Total absolute error: " << error << endl;

    return errors;
}


void getGaussianFilter(thrust::host_vector<float>& filter, int sigma) {
    /* This function returns a vector containing a 1d gaussian filter assuming 0 mean.
     * Since we are assuming the filter is symmetric, this is a more
     * efficient approach instead of using a 2d gaussian filter.
     */
    int k = 6 * sigma + 1;
    float sig = (float) sigma;
    float x;

    filter.resize(k);
    for (int i = 0; i < filter.size(); i++) {
        x = (float) (i - k / 2);
        filter[i] = (float)(exp(-1.0 * x * x / (2.0 * sig * sig)) / sqrt(2.0 * sig * sig * M_PI));
        //filter[i] = 1.0 / (float)(256);
    }
}


void convolveHeight(thrust::host_vector<pixel>& out, const pixel* sig, const float* filter, short& width, short& height, int filterSize) {
    short newHeight = height - filterSize + 1;
    float resultR;
    float resultG;
    float resultB;

    for (int i = 0; i < width; i++) {
        for (int j = 0; j < newHeight; j++) {
            resultR = 0.0;
            resultG = 0.0;
            resultB = 0.0;
            for (int k = 0; k < filterSize; k++) {
                resultR += sig[i + (j + k) * width].r * filter[k];
                resultG += sig[i + (j + k) * width].g * filter[k];
                resultB += sig[i + (j + k) * width].b * filter[k];
            }
            out[i + j * width].r = resultR;
            out[i + j * width].g = resultG;
            out[i + j * width].b = resultB;
        }
    }
    height = newHeight;
}


void convolveWidth(thrust::host_vector<pixel>& out, const pixel* sig, const float* filter, short& width, short& height, int filterSize) {
    short newWidth = width - filterSize + 1;
    float resultR;
    float resultG;
    float resultB;

    for (int i = 0; i < height; i++) {
        for (int j = 0; j < newWidth; j++) {
            resultR = 0.0;
            resultG = 0.0;
            resultB = 0.0;
            for (int k = 0; k < filterSize; k++) {
                resultR += sig[i * width + j + k].r * filter[k];
                resultG += sig[i * width + j + k].g * filter[k];
                resultB += sig[i * width + j + k].b * filter[k];
            }
            out[i * newWidth + j].r = resultR;
            out[i * newWidth + j].g = resultG;
            out[i * newWidth + j].b = resultB;
        }
    }
    width = newWidth;
}


////************************************** GPU implementations ********************************************
__global__
void convolveHeightKernel(pixel* out, const pixel* sig, const float* filter, const short width,
                          const short height, const short newHeight, const int filterSize) {
    extern __shared__ unsigned char sharedPtr[];                                // pointer to the shared memory
    float* filterShared = (float*)sharedPtr;                                  // ptr to kernel shared memory
    size_t ix = blockDim.x * blockIdx.x + threadIdx.x;
    size_t iy = blockDim.y * blockIdx.y + threadIdx.y;
    float r = 0.0;
    float g = 0.0;
    float b = 0.0;
    size_t idx;

    if ((ix < width) && (iy < newHeight)) {

        // copy filter to the shared memory
        for (int i = (threadIdx.y * blockDim.x + threadIdx.x); i < filterSize; i += blockDim.x * blockDim.y)
            filterShared[i] = filter[i];
        __syncthreads();

        for (int k = 0; k < filterSize; k++) {
            idx = ix + (iy + k) * width;
            r += sig[idx].r * filter[k];
            g += sig[idx].g * filter[k];
            b += sig[idx].b * filter[k];
        }
        idx = ix + iy * width;
        out[idx].r = r;
        out[idx].g = g;
        out[idx].b = b;
    }
}

__global__
void convolveWidthKernel(pixel* out, const pixel* sig, const float* filter, const short width,
                         const short newWidth, const short height, const int filterSize) {
    extern __shared__ unsigned char sharedPtr[];                                // pointer to the shared memory
    float* filterShared = (float*)sharedPtr;                                  // ptr to kernel shared memory
    size_t ix = blockDim.x * blockIdx.x + threadIdx.x;                          // calculate index along x dimension
    size_t iy = blockDim.y * blockIdx.y + threadIdx.y;                          // calculate index along y dimension
    size_t idx;                                                                 // temporary variable to hold index

    float r = 0.0;
    float g = 0.0;
    float b = 0.0;

    // check if the indices are out scope
    if ((ix < newWidth) && (iy < height)) {

        // copy filter to the shared memory
        for (int i = (threadIdx.y * blockDim.x + threadIdx.x); i < filterSize; i += blockDim.x * blockDim.y)
            filterShared[i] = filter[i];
        __syncthreads();

        // perform convolution
        for (int k = 0; k < filterSize; k++) {
            idx = iy * width + ix + k;
            r += sig[idx].r * filter[k];
            g += sig[idx].g * filter[k];
            b += sig[idx].b * filter[k];
        }

        // put data into the output
        idx = iy * newWidth + ix;
        out[idx].r = r;
        out[idx].g = g;
        out[idx].b = b;
    }
}

thrust::host_vector<pixel> convolve2dGpu(const thrust::host_vector<pixel> sig, const thrust::host_vector<float>filter,
                                         short& width, short& height, const int filterSize, FileHandler& handler){
    short newWidth, newHeight;

    ////********************************** perform convolution along the width ****************************
    //// define device variables for convolution along the width
    newWidth = width - filterSize + 1;
    thrust::device_vector<pixel> out1Gpu(newWidth * height);
    pixel* out1Ptr = thrust::raw_pointer_cast(out1Gpu.data());

    thrust::device_vector<pixel> in1 = sig;
    pixel* in1Ptr = thrust::raw_pointer_cast(in1.data());

    thrust::device_vector<float> convFilterGpu = filter;
    float* filterPtr = thrust::raw_pointer_cast(convFilterGpu.data());

    //// define filter launch parameters for convolution along the width
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, 0);
    dim3 blockDimension(32, 16, 1);
    dim3 gridDimension((newWidth + blockDimension.x - 1) / blockDimension.x,
                       (height + blockDimension.y - 1) / blockDimension.y, 1);

    // shared memory size
    size_t sharedBytes = filterSize * sizeof(float);

    convolveWidthKernel<<<gridDimension, blockDimension, sharedBytes>>>
                            (out1Ptr, in1Ptr, filterPtr, width, newWidth, height, filterSize);
    hipDeviceSynchronize();
    width = newWidth;

    ////********************************** perform convolution along the height ***************************
    newHeight = height - filterSize + 1;
    gridDimension.y = (newHeight + blockDimension.y - 1) / blockDimension.y;

    thrust::device_vector<pixel> out2Gpu(newWidth * newHeight);
    pixel* out2Ptr = thrust::raw_pointer_cast(out2Gpu.data());
    convolveHeightKernel<<<gridDimension, blockDimension, sharedBytes>>>
                            (out2Ptr, out1Ptr, filterPtr, width, height, newHeight, filterSize);
    hipDeviceSynchronize();
    height = newHeight;

    thrust::host_vector<pixel> outHost = out2Gpu;

    return outHost;
}


int main(int argc, char* argv[]) {
    ////********************************** parse command line arguments **********************************
    if (argc != 3)
        return 1;
    string filename(argv[1]);
    int sigma = atoi(argv[2]);

    ////********************************** declare variables *********************************************
    FileHandler handler;
    short width;
    short height;

    ////********************************** generate gaussian filter **************************************
    thrust::host_vector<float> filter;
    getGaussianFilter(filter, sigma);
    float* kernelPtr = &filter[0];

    ////********************************** load image ****************************************************
    thrust::host_vector<char> imageRaw = handler.loadImage(filename, width, height);
    const short widthInit = width;
    const short heightInit = height;
    Image image(imageRaw, width, height);
    pixel* imagePtr = image.getPointer();

    //////********************************** perform convolution(CPU) **************************************
    auto start = timer::now();
    thrust::host_vector<pixel> out1Cpu;
    out1Cpu.resize((width - filter.size() + 1) * height);
    convolveWidth(out1Cpu, imagePtr, kernelPtr, width, height, filter.size());

    thrust::host_vector<pixel> out2Cpu;
    out2Cpu.resize(width * (height - filter.size() + 1));
    convolveHeight(out2Cpu, &out1Cpu[0], kernelPtr, width, height, filter.size());
    auto end = timer::now();

    // report computation time
    std::chrono::milliseconds t = std::chrono::duration_cast<std::chrono::milliseconds> (end - start);
    cout << "CPU Elapsed time: " << t.count() << "ms" << endl;

    //////********************************** save data (CPU) *********************************************
    Image outputCpu(out2Cpu, width, height);
    thrust::host_vector<char> bytesCpu;
    outputCpu.toBytes(bytesCpu, true);
    handler.saveImage(bytesCpu, "./resultCpu.tga", width, height);

    ////********************************** perform convolution(GPU) **************************************
    width = widthInit;
    height = heightInit;
    start = timer::now();
    thrust::host_vector<pixel> outGpu = convolve2dGpu(image.getImage(), filter, width, height, filter.size(), handler) ;
    end = timer::now();

    t = std::chrono::duration_cast<std::chrono::milliseconds> (end - start);
    cout << "GPU Elapsed time: " << t.count() << "ms" << endl;

    //////********************************** save data (GPU) *********************************************
    Image outputGpu(outGpu, width, height);
    thrust::host_vector<char> bytesGpu;
    outputGpu.toBytes(bytesGpu, true);
    handler.saveImage(bytesGpu, "./resultGpu.tga", width, height);

    //////********************************** save data (err) *********************************************
    thrust::host_vector<pixel> errors = checkCalcError(out2Cpu, outGpu);
    return 0;
}
